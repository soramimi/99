#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(unsigned char *p)
{
//	int i = blockIdx.x;
//	int j = blockIdx.y;
	int i = threadIdx.x;
	int j = threadIdx.y;
	p[i * 9 + j] = (i + 1) * (j + 1);
}

int main( void )
{
	int devcount = 0;
	hipError_t error = hipGetDeviceCount(&devcount);
	if (error != hipSuccess || devcount < 1) {
		fprintf(stderr, "CUDA device not found\n");
	}

	unsigned char table[81];
	unsigned char *mem;
	hipMalloc((void **)&mem, 81);
//	dim3 b(9, 9);
	dim3 t(9, 9);
	kernel<<<1,t>>>(mem);
	hipMemcpy(table, mem, 81, hipMemcpyDeviceToHost);
	hipFree(mem);

	for (int i = 0; i < 9; i++) {
		for (int j = 0; j < 9; j++) {
			printf(" %2d", table[i * 9 + j]);
		}
		putchar('\n');
	}

	return 0;
}
